
#include <hip/hip_runtime.h>
// Copyright (c)       2017 Madhavan Seshadri
//                     2018 Patrick Diehl
//
// Distributed under the Boost Software License, Version 1.0. (See accompanying
// file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)
extern "C" { __global__ void kernel(char *out, int *width, int *height, int *yStart, int* n){
	unsigned int xDim = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yDim = blockIdx.y * blockDim.y + threadIdx.y;

	//index of the output array, multiplied by 3 for R,G,B values
	int arrayIndex = 3 * (*width) * yDim + xDim*3;
    
	float xPoint = ((float) (xDim)/(*width)) * 3.25f - 2.0f;
	float yPoint = ((float) (yDim+*yStart)/(*height)) * 2.5f - 1.25f; 

	//for calculation of complex number
	float x = 0.0;
	float y = 0.0;

	int iterationCount = 0;
	int numIterations = 256;
	//terminating condition x^2+y^2 < 4 or iterations >= numIterations
	while(y*y+x*x<=4 && iterationCount<(numIterations)){
		float xTemp = x*x-y*y + xPoint;
		y = 2*x*y + yPoint;
		x = xTemp;
		iterationCount++;
	}
    
    if (arrayIndex < *n)
    { 
	if(iterationCount == (numIterations)){
		out[arrayIndex] = iterationCount;
		out[arrayIndex+1]=1;
		out[arrayIndex+2]=iterationCount;
	}else{
		out[arrayIndex] = 0;
		out[arrayIndex+1]=iterationCount;
		out[arrayIndex+2]=0;
	}
   }
 }
};
